#include "hip/hip_runtime.h"
/*
 * Copyright 2021, Blender Foundation.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* Copy of the regular kernels with additional shader ray-tracing kernel that takes
 * much longer to compiler. This is only loaded when needed by the scene. */

#include "kernel/device/optix/kernel.cu"
#include "kernel/integrator/integrator_shade_surface.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface_raytrace()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (__params.path_index_array) ? __params.path_index_array[global_index] :
                                                       global_index;
  integrator_shade_surface_raytrace(nullptr, path_index, __params.render_buffer);
}
