#include "hip/hip_runtime.h"
/*
 * Copyright 2019, NVIDIA Corporation.
 * Copyright 2019, Blender Foundation.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// clang-format off
#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/device/gpu/image.h"  // Texture lookup uses normal CUDA intrinsics

#include "kernel/integrator/integrator_state.h"
#include "kernel/integrator/integrator_state_flow.h"
#include "kernel/integrator/integrator_state_util.h"

#include "kernel/integrator/integrator_intersect_closest.h"
#include "kernel/integrator/integrator_intersect_shadow.h"
#include "kernel/integrator/integrator_intersect_subsurface.h"
#include "kernel/integrator/integrator_intersect_volume_stack.h"

// clang-format on

template<typename T> ccl_device_forceinline T *get_payload_ptr_0()
{
  return (T *)(((uint64_t)optixGetPayload_1() << 32) | optixGetPayload_0());
}
template<typename T> ccl_device_forceinline T *get_payload_ptr_2()
{
  return (T *)(((uint64_t)optixGetPayload_3() << 32) | optixGetPayload_2());
}

template<bool always = false> ccl_device_forceinline uint get_object_id()
{
#ifdef __OBJECT_MOTION__
  // Always get the the instance ID from the TLAS
  // There might be a motion transform node between TLAS and BLAS which does not have one
  uint object = optixGetInstanceIdFromHandle(optixGetTransformListHandle(0));
#else
  uint object = optixGetInstanceId();
#endif
  // Choose between always returning object ID or only for instances
  if (always || (object & 1) == 0)
    // Can just remove the low bit since instance always contains object ID
    return object >> 1;
  else
    // Set to OBJECT_NONE if this is not an instanced object
    return OBJECT_NONE;
}

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_closest()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (__params.path_index_array) ? __params.path_index_array[global_index] :
                                                       global_index;
  integrator_intersect_closest(nullptr, path_index);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_shadow()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (__params.path_index_array) ? __params.path_index_array[global_index] :
                                                       global_index;
  integrator_intersect_shadow(nullptr, path_index);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_subsurface()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (__params.path_index_array) ? __params.path_index_array[global_index] :
                                                       global_index;
  integrator_intersect_subsurface(nullptr, path_index);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_volume_stack()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (__params.path_index_array) ? __params.path_index_array[global_index] :
                                                       global_index;
  integrator_intersect_volume_stack(nullptr, path_index);
}

extern "C" __global__ void __miss__kernel_optix_miss()
{
  // 'kernel_path_lamp_emission' checks intersection distance, so need to set it even on a miss
  optixSetPayload_0(__float_as_uint(optixGetRayTmax()));
  optixSetPayload_5(PRIMITIVE_NONE);
}

extern "C" __global__ void __anyhit__kernel_optix_local_hit()
{
#ifdef __BVH_LOCAL__
  const uint object = get_object_id<true>();
  if (object != optixGetPayload_4() /* local_object */) {
    // Only intersect with matching object
    return optixIgnoreIntersection();
  }

  const uint max_hits = optixGetPayload_5();
  if (max_hits == 0) {
    // Special case for when no hit information is requested, just report that something was hit
    optixSetPayload_5(true);
    return optixTerminateRay();
  }

  int hit = 0;
  uint *const lcg_state = get_payload_ptr_0<uint>();
  LocalIntersection *const local_isect = get_payload_ptr_2<LocalIntersection>();

  if (lcg_state) {
    for (int i = min(max_hits, local_isect->num_hits) - 1; i >= 0; --i) {
      if (optixGetRayTmax() == local_isect->hits[i].t) {
        return optixIgnoreIntersection();
      }
    }

    hit = local_isect->num_hits++;

    if (local_isect->num_hits > max_hits) {
      hit = lcg_step_uint(lcg_state) % local_isect->num_hits;
      if (hit >= max_hits) {
        return optixIgnoreIntersection();
      }
    }
  }
  else {
    if (local_isect->num_hits && optixGetRayTmax() > local_isect->hits[0].t) {
      // Record closest intersection only
      // Do not terminate ray here, since there is no guarantee about distance ordering in any-hit
      return optixIgnoreIntersection();
    }

    local_isect->num_hits = 1;
  }

  Intersection *isect = &local_isect->hits[hit];
  isect->t = optixGetRayTmax();
  isect->prim = optixGetPrimitiveIndex();
  isect->object = get_object_id();
  isect->type = kernel_tex_fetch(__prim_type, isect->prim);

  const float2 barycentrics = optixGetTriangleBarycentrics();
  isect->u = 1.0f - barycentrics.y - barycentrics.x;
  isect->v = barycentrics.x;

  // Record geometric normal
  const uint tri_vindex = kernel_tex_fetch(__prim_tri_index, isect->prim);
  const float3 tri_a = float4_to_float3(kernel_tex_fetch(__prim_tri_verts, tri_vindex + 0));
  const float3 tri_b = float4_to_float3(kernel_tex_fetch(__prim_tri_verts, tri_vindex + 1));
  const float3 tri_c = float4_to_float3(kernel_tex_fetch(__prim_tri_verts, tri_vindex + 2));
  local_isect->Ng[hit] = normalize(cross(tri_b - tri_a, tri_c - tri_a));

  // Continue tracing (without this the trace call would return after the first hit)
  optixIgnoreIntersection();
#endif
}

extern "C" __global__ void __anyhit__kernel_optix_shadow_all_hit()
{
#ifdef __SHADOW_RECORD_ALL__
  bool ignore_intersection = false;

  const uint prim = optixGetPrimitiveIndex();
#  ifdef __VISIBILITY_FLAG__
  const uint visibility = optixGetPayload_4();
  if ((kernel_tex_fetch(__prim_visibility, prim) & visibility) == 0) {
    ignore_intersection = true;
  }
#  endif

  float u = 0.0f, v = 0.0f;
  if (optixIsTriangleHit()) {
    const float2 barycentrics = optixGetTriangleBarycentrics();
    u = 1.0f - barycentrics.y - barycentrics.x;
    v = barycentrics.x;
  }
#  ifdef __HAIR__
  else {
    u = __uint_as_float(optixGetAttribute_0());
    v = __uint_as_float(optixGetAttribute_1());

    // Filter out curve endcaps
    if (u == 0.0f || u == 1.0f) {
      ignore_intersection = true;
    }
  }
#  endif

  int num_hits = optixGetPayload_2();
  int record_index = num_hits;
  const int max_hits = optixGetPayload_3();

  if (!ignore_intersection) {
    optixSetPayload_2(num_hits + 1);
  }

  Intersection *const isect_array = get_payload_ptr_0<Intersection>();

#  ifdef __TRANSPARENT_SHADOWS__
  if (num_hits >= max_hits) {
    /* If maximum number of hits reached, find a hit to replace. */
    const int num_recorded_hits = min(max_hits, num_hits);
    float max_recorded_t = isect_array[0].t;
    int max_recorded_hit = 0;

    for (int i = 1; i < num_recorded_hits; i++) {
      if (isect_array[i].t > max_recorded_t) {
        max_recorded_t = isect_array[i].t;
        max_recorded_hit = i;
      }
    }

    if (optixGetRayTmax() >= max_recorded_t) {
      /* Accept hit, so that OptiX won't consider any more hits beyond the distance of the current
       * hit anymore. */
      return;
    }

    record_index = max_recorded_hit;
  }
#  endif

  if (!ignore_intersection) {
    Intersection *const isect = isect_array + record_index;
    isect->u = u;
    isect->v = v;
    isect->t = optixGetRayTmax();
    isect->prim = prim;
    isect->object = get_object_id();
    isect->type = kernel_tex_fetch(__prim_type, prim);

#  ifdef __TRANSPARENT_SHADOWS__
    // Detect if this surface has a shader with transparent shadows
    if (!shader_transparent_shadow(NULL, isect) || max_hits == 0) {
#  endif
      // If no transparent shadows, all light is blocked and we can stop immediately
      optixSetPayload_5(true);
      return optixTerminateRay();
#  ifdef __TRANSPARENT_SHADOWS__
    }
#  endif
  }

  // Continue tracing
  optixIgnoreIntersection();
#endif
}

extern "C" __global__ void __anyhit__kernel_optix_visibility_test()
{
  uint visibility = optixGetPayload_4();
#ifdef __VISIBILITY_FLAG__
  const uint prim = optixGetPrimitiveIndex();
  if ((kernel_tex_fetch(__prim_visibility, prim) & visibility) == 0) {
    return optixIgnoreIntersection();
  }
#endif

#ifdef __HAIR__
  if (!optixIsTriangleHit()) {
    // Filter out curve endcaps
    const float u = __uint_as_float(optixGetAttribute_0());
    if (u == 0.0f || u == 1.0f) {
      return optixIgnoreIntersection();
    }
  }
#endif

  // Shadow ray early termination
  if (visibility & PATH_RAY_SHADOW_OPAQUE) {
    return optixTerminateRay();
  }
}

extern "C" __global__ void __closesthit__kernel_optix_hit()
{
  optixSetPayload_0(__float_as_uint(optixGetRayTmax()));  // Intersection distance
  optixSetPayload_3(optixGetPrimitiveIndex());
  optixSetPayload_4(get_object_id());
  // Can be PRIMITIVE_TRIANGLE and PRIMITIVE_MOTION_TRIANGLE or curve type and segment index
  optixSetPayload_5(kernel_tex_fetch(__prim_type, optixGetPrimitiveIndex()));

  if (optixIsTriangleHit()) {
    const float2 barycentrics = optixGetTriangleBarycentrics();
    optixSetPayload_1(__float_as_uint(1.0f - barycentrics.y - barycentrics.x));
    optixSetPayload_2(__float_as_uint(barycentrics.x));
  }
  else {
    optixSetPayload_1(optixGetAttribute_0());  // Same as 'optixGetCurveParameter()'
    optixSetPayload_2(optixGetAttribute_1());
  }
}

#ifdef __HAIR__
ccl_device_inline void optix_intersection_curve(const uint prim, const uint type)
{
  const uint object = get_object_id<true>();
  const uint visibility = optixGetPayload_4();

  float3 P = optixGetObjectRayOrigin();
  float3 dir = optixGetObjectRayDirection();

  // The direction is not normalized by default, but the curve intersection routine expects that
  float len;
  dir = normalize_len(dir, &len);

#  ifdef __OBJECT_MOTION__
  const float time = optixGetRayTime();
#  else
  const float time = 0.0f;
#  endif

  Intersection isect;
  isect.t = optixGetRayTmax();
  // Transform maximum distance into object space
  if (isect.t != FLT_MAX)
    isect.t *= len;

  if (curve_intersect(NULL, &isect, P, dir, isect.t, visibility, object, prim, time, type)) {
    optixReportIntersection(isect.t / len,
                            type & PRIMITIVE_ALL,
                            __float_as_int(isect.u),   // Attribute_0
                            __float_as_int(isect.v));  // Attribute_1
  }
}

extern "C" __global__ void __intersection__curve_ribbon()
{
  const uint prim = optixGetPrimitiveIndex();
  const uint type = kernel_tex_fetch(__prim_type, prim);

  if (type & (PRIMITIVE_CURVE_RIBBON | PRIMITIVE_MOTION_CURVE_RIBBON)) {
    optix_intersection_curve(prim, type);
  }
}
#endif
