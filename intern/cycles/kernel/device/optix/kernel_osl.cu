#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#define WITH_OSL

/* Copy of the regular OptiX kernels with additional OSL support. */

#include "kernel/device/optix/kernel_shader_raytrace.cu"

#include "kernel/bake/bake.h"
#include "kernel/integrator/init_from_camera.h"
#include "kernel/integrator/shade_background.h"
#include "kernel/integrator/shade_dedicated_light.h"
#include "kernel/integrator/shade_light.h"
#include "kernel/integrator/shade_shadow.h"
#include "kernel/integrator/shade_volume.h"

#include "kernel/device/gpu/work_stealing.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_background()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_background(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_light()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_light(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_surface(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_volume()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_volume(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_shadow()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_shadow(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_dedicated_light()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_dedicated_light(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_displace()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_displace_evaluate(nullptr, input, output, global_index);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_background()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_background_evaluate(nullptr, input, output, global_index);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_curve_shadow_transparency()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_curve_shadow_transparency_evaluate(nullptr, input, output, global_index);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_init_from_camera()
{
  const int global_index = optixGetLaunchIndex().x;

  const KernelWorkTile *tiles = (const KernelWorkTile *)kernel_params.path_index_array;

  const int tile_index = global_index / kernel_params.max_tile_work_size;
  const int tile_work_index = global_index - tile_index * kernel_params.max_tile_work_size;

  const KernelWorkTile *tile = &tiles[tile_index];

  if (tile_work_index >= tile->work_size) {
    return;
  }

  const int path_index = tile->path_index_offset + tile_work_index;

  uint x, y, sample;
  get_work_pixel(tile, tile_work_index, &x, &y, &sample);

  integrator_init_from_camera(nullptr, path_index, tile, kernel_params.render_buffer, x, y, sample);
}
