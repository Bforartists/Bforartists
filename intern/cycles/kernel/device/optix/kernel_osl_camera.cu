#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2011-2025 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#define WITH_OSL

#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/integrator/init_from_camera.h"

#include "kernel/device/gpu/work_stealing.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_init_from_camera()
{
  const int global_index = optixGetLaunchIndex().x;

  const KernelWorkTile *tiles = (const KernelWorkTile *)kernel_params.path_index_array;

  const int tile_index = global_index / kernel_params.max_tile_work_size;
  const int tile_work_index = global_index - tile_index * kernel_params.max_tile_work_size;

  const KernelWorkTile *tile = &tiles[tile_index];

  if (tile_work_index >= tile->work_size) {
    return;
  }

  const int path_index = tile->path_index_offset + tile_work_index;

  uint x, y, sample;
  get_work_pixel(tile, tile_work_index, &x, &y, &sample);

  integrator_init_from_camera(nullptr, path_index, tile, kernel_params.render_buffer, x, y, sample);
}
