#include "hip/hip_runtime.h"
/*
 * Copyright 2019, NVIDIA Corporation.
 * Copyright 2019, Blender Foundation.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernel/kernel_compat_optix.h"
#include "util/util_atomic.h"
#include "kernel/kernel_types.h"
#include "kernel/kernel_globals.h"
#include "../cuda/kernel_cuda_image.h"  // Texture lookup uses normal CUDA intrinsics

#include "kernel/kernel_path.h"
#include "kernel/kernel_bake.h"

template<typename T> ccl_device_forceinline T *get_payload_ptr_0()
{
  return (T *)(((uint64_t)optixGetPayload_1() << 32) | optixGetPayload_0());
}
template<typename T> ccl_device_forceinline T *get_payload_ptr_2()
{
  return (T *)(((uint64_t)optixGetPayload_3() << 32) | optixGetPayload_2());
}

template<bool always = false> ccl_device_forceinline uint get_object_id()
{
#ifdef __OBJECT_MOTION__
  // Always get the the instance ID from the TLAS
  // There might be a motion transform node between TLAS and BLAS which does not have one
  uint object = optixGetInstanceIdFromHandle(optixGetTransformListHandle(0));
#else
  uint object = optixGetInstanceId();
#endif
  // Choose between always returning object ID or only for instances
  if (always)
    // Can just remove the high bit since instance always contains object ID
    return object & 0x7FFFFF;
  // Set to OBJECT_NONE if this is not an instanced object
  else if (object & 0x800000)
    object = OBJECT_NONE;
  return object;
}

extern "C" __global__ void __raygen__kernel_optix_path_trace()
{
  KernelGlobals kg;  // Allocate stack storage for common data

  const uint3 launch_index = optixGetLaunchIndex();
  // Keep threads for same pixel together to improve occupancy of warps
  uint pixel_offset = launch_index.x / __params.tile.num_samples;
  uint sample_offset = launch_index.x % __params.tile.num_samples;

  kernel_path_trace(&kg,
                    __params.tile.buffer,
                    __params.tile.start_sample + sample_offset,
                    __params.tile.x + pixel_offset,
                    __params.tile.y + launch_index.y,
                    __params.tile.offset,
                    __params.tile.stride);
}

#ifdef __BAKING__
extern "C" __global__ void __raygen__kernel_optix_bake()
{
  KernelGlobals kg;
  const ShaderParams &p = __params.shader;
  kernel_bake_evaluate(&kg,
                       p.input,
                       p.output,
                       (ShaderEvalType)p.type,
                       p.filter,
                       p.sx + optixGetLaunchIndex().x,
                       p.offset,
                       p.sample);
}
#endif

extern "C" __global__ void __raygen__kernel_optix_displace()
{
  KernelGlobals kg;
  const ShaderParams &p = __params.shader;
  kernel_displace_evaluate(&kg, p.input, p.output, p.sx + optixGetLaunchIndex().x);
}

extern "C" __global__ void __raygen__kernel_optix_background()
{
  KernelGlobals kg;
  const ShaderParams &p = __params.shader;
  kernel_background_evaluate(&kg, p.input, p.output, p.sx + optixGetLaunchIndex().x);
}

extern "C" __global__ void __miss__kernel_optix_miss()
{
  // 'kernel_path_lamp_emission' checks intersection distance, so need to set it even on a miss
  optixSetPayload_0(__float_as_uint(optixGetRayTmax()));
  optixSetPayload_5(PRIMITIVE_NONE);
}

extern "C" __global__ void __anyhit__kernel_optix_local_hit()
{
#ifdef __BVH_LOCAL__
  const uint object = get_object_id<true>();
  if (object != optixGetPayload_4() /* local_object */) {
    // Only intersect with matching object
    return optixIgnoreIntersection();
  }

  int hit = 0;
  uint *const lcg_state = get_payload_ptr_0<uint>();
  LocalIntersection *const local_isect = get_payload_ptr_2<LocalIntersection>();

  if (lcg_state) {
    const uint max_hits = optixGetPayload_5();
    for (int i = min(max_hits, local_isect->num_hits) - 1; i >= 0; --i) {
      if (optixGetRayTmax() == local_isect->hits[i].t) {
        return optixIgnoreIntersection();
      }
    }

    hit = local_isect->num_hits++;

    if (local_isect->num_hits > max_hits) {
      hit = lcg_step_uint(lcg_state) % local_isect->num_hits;
      if (hit >= max_hits) {
        return optixIgnoreIntersection();
      }
    }
  }
  else {
    if (local_isect->num_hits && optixGetRayTmax() > local_isect->hits[0].t) {
      // Record closest intersection only (do not terminate ray here, since there is no guarantee
      // about distance ordering in anyhit)
      return optixIgnoreIntersection();
    }

    local_isect->num_hits = 1;
  }

  Intersection *isect = &local_isect->hits[hit];
  isect->t = optixGetRayTmax();
  isect->prim = optixGetPrimitiveIndex();
  isect->object = get_object_id();
  isect->type = kernel_tex_fetch(__prim_type, isect->prim);

  if (optixIsTriangleHit()) {
    const float2 barycentrics = optixGetTriangleBarycentrics();
    isect->u = 1.0f - barycentrics.y - barycentrics.x;
    isect->v = barycentrics.x;
  }
  else {
    isect->u = __uint_as_float(optixGetAttribute_0());
    isect->v = __uint_as_float(optixGetAttribute_1());
  }

  // Record geometric normal
  const uint tri_vindex = kernel_tex_fetch(__prim_tri_index, isect->prim);
  const float3 tri_a = float4_to_float3(kernel_tex_fetch(__prim_tri_verts, tri_vindex + 0));
  const float3 tri_b = float4_to_float3(kernel_tex_fetch(__prim_tri_verts, tri_vindex + 1));
  const float3 tri_c = float4_to_float3(kernel_tex_fetch(__prim_tri_verts, tri_vindex + 2));
  local_isect->Ng[hit] = normalize(cross(tri_b - tri_a, tri_c - tri_a));

  // Continue tracing (without this the trace call would return after the first hit)
  optixIgnoreIntersection();
#endif
}

extern "C" __global__ void __anyhit__kernel_optix_shadow_all_hit()
{
#ifdef __SHADOW_RECORD_ALL__
  const uint prim = optixGetPrimitiveIndex();
#  ifdef __VISIBILITY_FLAG__
  const uint visibility = optixGetPayload_4();
  if ((kernel_tex_fetch(__prim_visibility, prim) & visibility) == 0) {
    return optixIgnoreIntersection();
  }
#  endif

  // Offset into array with num_hits
  Intersection *const isect = get_payload_ptr_0<Intersection>() + optixGetPayload_2();
  isect->t = optixGetRayTmax();
  isect->prim = prim;
  isect->object = get_object_id();
  isect->type = kernel_tex_fetch(__prim_type, prim);

  if (optixIsTriangleHit()) {
    const float2 barycentrics = optixGetTriangleBarycentrics();
    isect->u = 1.0f - barycentrics.y - barycentrics.x;
    isect->v = barycentrics.x;
  }
  else {
    isect->u = __uint_as_float(optixGetAttribute_0());
    isect->v = __uint_as_float(optixGetAttribute_1());
  }

#  ifdef __TRANSPARENT_SHADOWS__
  // Detect if this surface has a shader with transparent shadows
  if (!shader_transparent_shadow(NULL, isect) || optixGetPayload_2() >= optixGetPayload_3()) {
#  endif
    // This is an opaque hit or the hit limit has been reached, abort traversal
    optixSetPayload_5(true);
    return optixTerminateRay();
#  ifdef __TRANSPARENT_SHADOWS__
  }

  // TODO(pmours): Do we need REQUIRE_UNIQUE_ANYHIT for this to work?
  optixSetPayload_2(optixGetPayload_2() + 1);  // num_hits++

  // Continue tracing
  optixIgnoreIntersection();
#  endif
#endif
}

extern "C" __global__ void __anyhit__kernel_optix_visibility_test()
{
  uint visibility = optixGetPayload_4();
#ifdef __VISIBILITY_FLAG__
  const uint prim = optixGetPrimitiveIndex();
  if ((kernel_tex_fetch(__prim_visibility, prim) & visibility) == 0)
    return optixIgnoreIntersection();
#endif

  // Shadow ray early termination
  if (visibility & PATH_RAY_SHADOW_OPAQUE)
    return optixTerminateRay();
}

extern "C" __global__ void __closesthit__kernel_optix_hit()
{
  optixSetPayload_0(__float_as_uint(optixGetRayTmax()));  // Intersection distance
  optixSetPayload_3(optixGetPrimitiveIndex());
  optixSetPayload_4(get_object_id());
  // Can be PRIMITIVE_TRIANGLE and PRIMITIVE_MOTION_TRIANGLE or curve type and segment index
  optixSetPayload_5(kernel_tex_fetch(__prim_type, optixGetPrimitiveIndex()));

  if (optixIsTriangleHit()) {
    const float2 barycentrics = optixGetTriangleBarycentrics();
    optixSetPayload_1(__float_as_uint(1.0f - barycentrics.y - barycentrics.x));
    optixSetPayload_2(__float_as_uint(barycentrics.x));
  }
  else {
    optixSetPayload_1(optixGetAttribute_0());
    optixSetPayload_2(optixGetAttribute_1());
  }
}

#ifdef __HAIR__
ccl_device_inline void optix_intersection_curve(const uint prim, const uint type)
{
  const uint object = get_object_id<true>();
  const uint visibility = optixGetPayload_4();

  float3 P = optixGetObjectRayOrigin();
  float3 dir = optixGetObjectRayDirection();

  // The direction is not normalized by default, but the curve intersection routine expects that
  float len;
  dir = normalize_len(dir, &len);

#  ifdef __OBJECT_MOTION__
  const float time = optixGetRayTime();
#  else
  const float time = 0.0f;
#  endif

  Intersection isect;
  isect.t = optixGetRayTmax();
  // Transform maximum distance into object space
  if (isect.t != FLT_MAX)
    isect.t *= len;

  if (curve_intersect(NULL, &isect, P, dir, visibility, object, prim, time, type)) {
    optixReportIntersection(isect.t / len,
                            type & PRIMITIVE_ALL,
                            __float_as_int(isect.u),   // Attribute_0
                            __float_as_int(isect.v));  // Attribute_1
  }

}

extern "C" __global__ void __intersection__curve_ribbon()
{
  const uint prim = optixGetPrimitiveIndex();
  const uint type = kernel_tex_fetch(__prim_type, prim);

  if (type & (PRIMITIVE_CURVE_RIBBON | PRIMITIVE_MOTION_CURVE_RIBBON)) {
    optix_intersection_curve(prim, type);
  }
}

extern "C" __global__ void __intersection__curve_all()
{
  const uint prim = optixGetPrimitiveIndex();
  const uint type = kernel_tex_fetch(__prim_type, prim);
  optix_intersection_curve(prim, type);
}
#endif

#ifdef __KERNEL_DEBUG__
extern "C" __global__ void __exception__kernel_optix_exception()
{
  printf("Unhandled exception occured: code %d!\n", optixGetExceptionCode());
}
#endif
