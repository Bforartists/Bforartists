
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "kernel/kernel_compat_cuda.h"
#include "kernel_config.h"

#include "util/util_atomic.h"

#include "kernel/kernel_math.h"
#include "kernel/kernel_types.h"
#include "kernel/kernel_globals.h"
#include "kernel/kernel_color.h"
#include "kernel/kernels/cuda/kernel_cuda_image.h"
#include "kernel/kernel_film.h"
#include "kernel/kernel_path.h"
#include "kernel/kernel_path_branched.h"
#include "kernel/kernel_bake.h"
#include "kernel/kernel_work_stealing.h"
#include "kernel/kernel_adaptive_sampling.h"

/* kernels */
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace(WorkTile *tile, uint total_work_size)
{
	int work_index = ccl_global_id(0);
	bool thread_is_active = work_index < total_work_size;
	uint x, y, sample;
	KernelGlobals kg;
	if(thread_is_active) {
		get_work_pixel(tile, work_index, &x, &y, &sample);

		kernel_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}

	if(kernel_data.film.cryptomatte_passes) {
		__syncthreads();
		if(thread_is_active) {
			kernel_cryptomatte_post(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
		}
	}
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_BRANCHED_MAX_REGISTERS)
kernel_cuda_branched_path_trace(WorkTile *tile, uint total_work_size)
{
	int work_index = ccl_global_id(0);
	bool thread_is_active = work_index < total_work_size;
	uint x, y, sample;
	KernelGlobals kg;
	if(thread_is_active) {
		get_work_pixel(tile, work_index, &x, &y, &sample);

		kernel_branched_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}
	
	if(kernel_data.film.cryptomatte_passes) {
		__syncthreads();
		if(thread_is_active) {
			kernel_cryptomatte_post(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
		}
	}
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_adaptive_stopping(WorkTile *tile, int sample, uint total_work_size)
{
	int work_index = ccl_global_id(0);
	bool thread_is_active = work_index < total_work_size;
	KernelGlobals kg;
	if(thread_is_active && kernel_data.film.pass_adaptive_aux_buffer) {
		uint x = tile->x + work_index % tile->w;
		uint y = tile->y + work_index / tile->w;
		int index = tile->offset + x + y * tile->stride;
		ccl_global float *buffer = tile->buffer + index * kernel_data.film.pass_stride;
		kernel_do_adaptive_stopping(&kg, buffer, sample);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_adaptive_filter_x(WorkTile *tile, int sample, uint)
{
	KernelGlobals kg;
	if(kernel_data.film.pass_adaptive_aux_buffer && sample > kernel_data.integrator.adaptive_min_samples) {
		if(ccl_global_id(0) < tile->h) {
			int y = tile->y + ccl_global_id(0);
			kernel_do_adaptive_filter_x(&kg, y, tile);
		}
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_adaptive_filter_y(WorkTile *tile, int sample, uint)
{
	KernelGlobals kg;
	if(kernel_data.film.pass_adaptive_aux_buffer && sample > kernel_data.integrator.adaptive_min_samples) {
		if(ccl_global_id(0) < tile->w) {
			int x = tile->x + ccl_global_id(0);
			kernel_do_adaptive_filter_y(&kg, x, tile);
		}
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_adaptive_scale_samples(WorkTile *tile, int start_sample, int sample, uint total_work_size)
{
	if(kernel_data.film.pass_adaptive_aux_buffer) {
		int work_index = ccl_global_id(0);
		bool thread_is_active = work_index < total_work_size;
		KernelGlobals kg;
		if(thread_is_active) {
			uint x = tile->x + work_index % tile->w;
			uint y = tile->y + work_index / tile->w;
			int index = tile->offset + x + y * tile->stride;
			ccl_global float *buffer = tile->buffer + index * kernel_data.film.pass_stride;
			if(buffer[kernel_data.film.pass_sample_count] < 0.0f) {
				buffer[kernel_data.film.pass_sample_count] = -buffer[kernel_data.film.pass_sample_count];
				float sample_multiplier = sample / max((float)start_sample + 1.0f, buffer[kernel_data.film.pass_sample_count]);
				if(sample_multiplier != 1.0f) {
					kernel_adaptive_post_adjust(&kg, buffer, sample_multiplier);
				}
			}
			else {
				kernel_adaptive_post_adjust(&kg, buffer, sample / (sample - 1.0f));
			}
		}
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_displace(uint4 *input,
                     float4 *output,
                     int type,
                     int sx,
                     int sw,
                     int offset,
                     int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_displace_evaluate(&kg, input, output, x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_background(uint4 *input,
                       float4 *output,
                       int type,
                       int sx,
                       int sw,
                       int offset,
                       int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_background_evaluate(&kg, input, output, x);
	}
}

#ifdef __BAKING__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_bake(WorkTile *tile, uint total_work_size)
{
	int work_index = ccl_global_id(0);

	if(work_index < total_work_size) {
		uint x, y, sample;
		get_work_pixel(tile, work_index, &x, &y, &sample);

		KernelGlobals kg;
		kernel_bake_evaluate(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}
}
#endif

#endif

