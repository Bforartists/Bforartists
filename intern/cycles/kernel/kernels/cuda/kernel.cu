#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#include "../../kernel_compat_cuda.h"
#include "../../kernel_math.h"
#include "../../kernel_types.h"
#include "../../kernel_globals.h"
#include "../../kernel_film.h"
#include "../../kernel_path.h"
#include "../../kernel_path_branched.h"
#include "../../kernel_bake.h"

/* device data taken from CUDA occupancy calculator */

#ifdef __CUDA_ARCH__

/* 2.0 and 2.1 */
#if __CUDA_ARCH__ == 200 || __CUDA_ARCH__ == 210
#define CUDA_MULTIPRESSOR_MAX_REGISTERS 32768
#define CUDA_MULTIPROCESSOR_MAX_BLOCKS 8
#define CUDA_BLOCK_MAX_THREADS 1024
#define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#define CUDA_THREADS_BLOCK_WIDTH 16
#define CUDA_KERNEL_MAX_REGISTERS 32
#define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 40

/* 3.0 and 3.5 */
#elif __CUDA_ARCH__ == 300 || __CUDA_ARCH__ == 350
#define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#define CUDA_BLOCK_MAX_THREADS 1024
#define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#define CUDA_THREADS_BLOCK_WIDTH 16
#define CUDA_KERNEL_MAX_REGISTERS 63
#define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 3.2 */
#elif __CUDA_ARCH__ == 320
#define CUDA_MULTIPRESSOR_MAX_REGISTERS 32768
#define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#define CUDA_BLOCK_MAX_THREADS 1024
#define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#define CUDA_THREADS_BLOCK_WIDTH 16
#define CUDA_KERNEL_MAX_REGISTERS 63
#define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 3.7 */
#elif __CUDA_ARCH__ == 370
#define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#define CUDA_BLOCK_MAX_THREADS 1024
#define CUDA_THREAD_MAX_REGISTERS 255

/* tunable parameters */
#define CUDA_THREADS_BLOCK_WIDTH 16
#define CUDA_KERNEL_MAX_REGISTERS 63
#define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 5.0, 5.2 and 5.3 */
#elif __CUDA_ARCH__ == 500 || __CUDA_ARCH__ == 520 || __CUDA_ARCH__ == 530
#define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#define CUDA_MULTIPROCESSOR_MAX_BLOCKS 32
#define CUDA_BLOCK_MAX_THREADS 1024
#define CUDA_THREAD_MAX_REGISTERS 255

/* tunable parameters */
#define CUDA_THREADS_BLOCK_WIDTH 16
#define CUDA_KERNEL_MAX_REGISTERS 40
#define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* unknown architecture */
#else
#error "Unknown or unsupported CUDA architecture, can't determine launch bounds"
#endif

/* compute number of threads per block and minimum blocks per multiprocessor
 * given the maximum number of registers per thread */

#define CUDA_LAUNCH_BOUNDS(threads_block_width, thread_num_registers) \
	__launch_bounds__( \
		threads_block_width*threads_block_width, \
		CUDA_MULTIPRESSOR_MAX_REGISTERS/(threads_block_width*threads_block_width*thread_num_registers) \
		)

/* sanity checks */

#if CUDA_THREADS_BLOCK_WIDTH*CUDA_THREADS_BLOCK_WIDTH > CUDA_BLOCK_MAX_THREADS
#error "Maximum number of threads per block exceeded"
#endif

#if CUDA_MULTIPRESSOR_MAX_REGISTERS/(CUDA_THREADS_BLOCK_WIDTH*CUDA_THREADS_BLOCK_WIDTH*CUDA_KERNEL_MAX_REGISTERS) > CUDA_MULTIPROCESSOR_MAX_BLOCKS
#error "Maximum number of blocks per multiprocessor exceeded"
#endif

#if CUDA_KERNEL_MAX_REGISTERS > CUDA_THREAD_MAX_REGISTERS
#error "Maximum number of registers per thread exceeded"
#endif

#if CUDA_KERNEL_BRANCHED_MAX_REGISTERS > CUDA_THREAD_MAX_REGISTERS
#error "Maximum number of registers per thread exceeded"
#endif

/* kernels */

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_BRANCHED_MAX_REGISTERS)
kernel_cuda_branched_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_branched_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_shader(uint4 *input,
                   float4 *output,
                   float *output_luma,
                   int type,
                   int sx,
                   int sw,
                   int offset,
                   int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		kernel_shader_evaluate(NULL,
		                       input,
		                       output,
		                       output_luma,
		                       (ShaderEvalType)type, 
		                       x,
		                       sample);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_bake(uint4 *input, float4 *output, int type, int filter, int sx, int sw, int offset, int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw)
		kernel_bake_evaluate(NULL, input, output, (ShaderEvalType)type, filter, x, offset, sample);
}

#endif

