
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "kernel/kernel_compat_cuda.h"
#include "kernel_config.h"

#include "util/util_atomic.h"

#include "kernel/kernel_math.h"
#include "kernel/kernel_types.h"
#include "kernel/kernel_globals.h"
#include "kernel/kernels/cuda/kernel_cuda_image.h"
#include "kernel/kernel_film.h"
#include "kernel/kernel_path.h"
#include "kernel/kernel_path_branched.h"
#include "kernel/kernel_bake.h"
#include "kernel/kernel_work_stealing.h"

/* kernels */
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace(WorkTile *tile, uint total_work_size)
{
	int work_index = ccl_global_id(0);

	if(work_index < total_work_size) {
		uint x, y, sample;
		get_work_pixel(tile, work_index, &x, &y, &sample);

		KernelGlobals kg;
		kernel_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_BRANCHED_MAX_REGISTERS)
kernel_cuda_branched_path_trace(WorkTile *tile, uint total_work_size)
{
	int work_index = ccl_global_id(0);

	if(work_index < total_work_size) {
		uint x, y, sample;
		get_work_pixel(tile, work_index, &x, &y, &sample);

		KernelGlobals kg;
		kernel_branched_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_displace(uint4 *input,
                     float4 *output,
                     int type,
                     int sx,
                     int sw,
                     int offset,
                     int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_displace_evaluate(&kg, input, output, x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_background(uint4 *input,
                       float4 *output,
                       int type,
                       int sx,
                       int sw,
                       int offset,
                       int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_background_evaluate(&kg, input, output, x);
	}
}

#ifdef __BAKING__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_bake(uint4 *input, float4 *output, int type, int filter, int sx, int sw, int offset, int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_bake_evaluate(&kg, input, output, (ShaderEvalType)type, filter, x, offset, sample);
	}
}
#endif

#endif

