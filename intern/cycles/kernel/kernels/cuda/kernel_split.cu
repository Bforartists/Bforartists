
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2016 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA split kernel entry points */

#ifdef __CUDA_ARCH__

#define __SPLIT_KERNEL__

#include "../../kernel_compat_cuda.h"
#include "kernel_config.h"

#include "../../split/kernel_split_common.h"
#include "../../split/kernel_data_init.h"
#include "../../split/kernel_path_init.h"
#include "../../split/kernel_scene_intersect.h"
#include "../../split/kernel_lamp_emission.h"
#include "../../split/kernel_queue_enqueue.h"
#include "../../split/kernel_background_buffer_update.h"
#include "../../split/kernel_shader_eval.h"
#include "../../split/kernel_holdout_emission_blurring_pathtermination_ao.h"
#include "../../split/kernel_direct_lighting.h"
#include "../../split/kernel_shadow_blocked.h"
#include "../../split/kernel_next_iteration_setup.h"

#include "../../kernel_film.h"

/* kernels */
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_state_buffer_size(uint num_threads, uint *size)
{
	*size = split_data_buffer_size(NULL, num_threads);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace_data_init(
        ccl_global void *split_data_buffer,
        int num_elements,
        ccl_global char *ray_state,
        ccl_global uint *rng_state,
        int start_sample,
        int end_sample,
        int sx, int sy, int sw, int sh, int offset, int stride,
        ccl_global int *Queue_index,
        int queuesize,
        ccl_global char *use_queues_flag,
        ccl_global unsigned int *work_pool_wgs,
        unsigned int num_samples,
        ccl_global float *buffer)
{
	kernel_data_init(NULL,
	                 NULL,
	                 split_data_buffer,
	                 num_elements,
	                 ray_state,
	                 rng_state,
	                 start_sample,
	                 end_sample,
	                 sx, sy, sw, sh, offset, stride,
	                 Queue_index,
	                 queuesize,
	                 use_queues_flag,
	                 work_pool_wgs,
	                 num_samples,
	                 buffer);
}

#define DEFINE_SPLIT_KERNEL_FUNCTION(name) \
	extern "C" __global__ void \
	CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS) \
	kernel_cuda_##name() \
	{ \
		kernel_##name(NULL); \
	}

DEFINE_SPLIT_KERNEL_FUNCTION(path_init)
DEFINE_SPLIT_KERNEL_FUNCTION(scene_intersect)
DEFINE_SPLIT_KERNEL_FUNCTION(lamp_emission)
DEFINE_SPLIT_KERNEL_FUNCTION(queue_enqueue)
DEFINE_SPLIT_KERNEL_FUNCTION(background_buffer_update)
DEFINE_SPLIT_KERNEL_FUNCTION(shader_eval)
DEFINE_SPLIT_KERNEL_FUNCTION(holdout_emission_blurring_pathtermination_ao)
DEFINE_SPLIT_KERNEL_FUNCTION(direct_lighting)
DEFINE_SPLIT_KERNEL_FUNCTION(shadow_blocked)
DEFINE_SPLIT_KERNEL_FUNCTION(next_iteration_setup)

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

#endif

