
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2017 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "kernel_config.h"

#include "kernel/kernel_compat_cuda.h"

#include "kernel/filter/filter_kernel.h"

/* kernels */

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_shadow(int sample,
                                 TilesInfo *tiles,
                                 float *unfilteredA,
                                 float *unfilteredB,
                                 float *sampleVariance,
                                 float *sampleVarianceV,
                                 float *bufferVariance,
                                 int4 prefilter_rect,
                                 int buffer_pass_stride,
                                 int buffer_denoising_offset)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_divide_shadow(sample,
		                            tiles,
		                            x, y,
		                            unfilteredA,
		                            unfilteredB,
		                            sampleVariance,
		                            sampleVarianceV,
		                            bufferVariance,
		                            prefilter_rect,
		                            buffer_pass_stride,
		                            buffer_denoising_offset);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_get_feature(int sample,
                               TilesInfo *tiles,
                               int m_offset,
                               int v_offset,
                               float *mean,
                               float *variance,
                               int4 prefilter_rect,
                               int buffer_pass_stride,
                               int buffer_denoising_offset)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_get_feature(sample,
		                          tiles,
		                          m_offset, v_offset,
		                          x, y,
		                          mean, variance,
		                          prefilter_rect,
		                          buffer_pass_stride,
		                          buffer_denoising_offset);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_detect_outliers(float *image,
                                   float *variance,
                                   float *depth,
                                   float *output,
                                   int4 prefilter_rect,
                                   int pass_stride)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_detect_outliers(x, y, image, variance, depth, output, prefilter_rect, pass_stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_combine_halves(float *mean, float *variance, float *a, float *b, int4 prefilter_rect, int r)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_combine_halves(x, y, mean, variance, a, b, prefilter_rect, r);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_construct_transform(float const* __restrict__ buffer,
                                       float *transform, int *rank,
                                       int4 filter_area, int4 rect,
                                       int radius, float pca_threshold,
                                       int pass_stride)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int *l_rank = rank + y*filter_area.z + x;
		float *l_transform = transform + y*filter_area.z + x;
		kernel_filter_construct_transform(buffer,
		                                  x + filter_area.x, y + filter_area.y,
		                                  rect, pass_stride,
		                                  l_transform, l_rank,
		                                  radius, pca_threshold,
		                                  filter_area.z*filter_area.w,
		                                  threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_difference(const float *ccl_restrict weight_image,
                                       const float *ccl_restrict variance_image,
                                       float *difference_image,
                                       int w,
                                       int h,
                                       int stride,
                                       int shift_stride,
                                       int r,
                                       int channel_offset,
                                       float a,
                                       float k_2)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, shift_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_calc_difference(co.x, co.y, co.z, co.w,
		                                  weight_image,
		                                  variance_image,
		                                  difference_image + ofs,
		                                  rect, stride,
		                                  channel_offset, a, k_2);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_blur(const float *ccl_restrict difference_image,
                            float *out_image,
                            int w,
                            int h,
                            int stride,
                            int shift_stride,
                            int r,
                            int f)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, shift_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_blur(co.x, co.y,
		                       difference_image + ofs,
		                       out_image + ofs,
		                       rect, stride, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_weight(const float *ccl_restrict difference_image,
                                   float *out_image,
                                   int w,
                                   int h,
                                   int stride,
                                   int shift_stride,
                                   int r,
                                   int f)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, shift_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_calc_weight(co.x, co.y,
		                              difference_image + ofs,
		                              out_image + ofs,
		                              rect, stride, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_update_output(const float *ccl_restrict difference_image,
                                     const float *ccl_restrict image,
                                     float *out_image,
                                     float *accum_image,
                                     int w,
                                     int h,
                                     int stride,
                                     int shift_stride,
                                     int r,
                                     int f)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, shift_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_update_output(co.x, co.y, co.z, co.w,
		                                difference_image + ofs,
		                                image,
		                                out_image,
		                                accum_image,
		                                rect, stride, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_normalize(float *out_image,
                                 const float *ccl_restrict accum_image,
                                 int w,
                                 int h,
                                 int stride)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < w && y < h) {
		kernel_filter_nlm_normalize(x, y, out_image, accum_image, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_construct_gramian(const float *ccl_restrict difference_image,
                                         const float *ccl_restrict buffer,
                                         float const* __restrict__ transform,
                                         int *rank,
                                         float *XtWX,
                                         float3 *XtWY,
                                         int4 filter_window,
                                         int w,
                                         int h,
                                         int stride,
                                         int shift_stride,
                                         int r,
                                         int f,
                                         int pass_stride)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords_window(w, h, r, shift_stride, &rect, &co, &ofs, filter_window)) {
		kernel_filter_nlm_construct_gramian(co.x, co.y,
		                                    co.z, co.w,
		                                    difference_image + ofs,
		                                    buffer,
		                                    transform, rank,
		                                    XtWX, XtWY,
		                                    rect, filter_window,
		                                    stride, f,
		                                    pass_stride,
		                                    threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_finalize(float *buffer,
                            int *rank,
                            float *XtWX,
                            float3 *XtWY,
                            int4 filter_area,
                            int4 buffer_params,
                            int sample)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int storage_ofs = y*filter_area.z+x;
		rank += storage_ofs;
		XtWX += storage_ofs;
		XtWY += storage_ofs;
		kernel_filter_finalize(x, y, buffer, rank,
		                       filter_area.z*filter_area.w,
		                       XtWX, XtWY,
		                       buffer_params, sample);
	}
}

#endif

