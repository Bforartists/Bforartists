
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2017 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "kernel_config.h"

#include "kernel/kernel_compat_cuda.h"

#include "kernel/filter/filter_kernel.h"

/* kernels */

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_copy_input(float *buffer,
                              CCL_FILTER_TILE_INFO,
                              int4 prefilter_rect,
                              int buffer_pass_stride)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		int xtile = (x < tile_info->x[1]) ? 0 : ((x < tile_info->x[2]) ? 1 : 2);
		int ytile = (y < tile_info->y[1]) ? 0 : ((y < tile_info->y[2]) ? 1 : 2);
		int itile = ytile * 3 + xtile;
		float *const in = ((float *)ccl_get_tile_buffer(itile)) +
			(tile_info->offsets[itile] + y * tile_info->strides[itile] + x) * buffer_pass_stride;
		buffer += ((y - prefilter_rect.y) * (prefilter_rect.z - prefilter_rect.x) + (x - prefilter_rect.x)) * buffer_pass_stride;
		for (int i = 0; i < buffer_pass_stride; ++i)
			buffer[i] = in[i];
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_convert_to_rgb(float *rgb, float *buf, int sw, int sh, int stride, int pass_stride, int3 pass_offset, int num_inputs, int num_samples)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < sw && y < sh) {
		if (num_inputs > 0) {
			float *in = buf + x * pass_stride + (y * stride + pass_offset.x) / sizeof(float);
			float *out = rgb + (x + y * sw) * 3;
			out[0] = clamp(in[0] / num_samples, 0.0f, 10000.0f);
			out[1] = clamp(in[1] / num_samples, 0.0f, 10000.0f);
			out[2] = clamp(in[2] / num_samples, 0.0f, 10000.0f);
		}
		if (num_inputs > 1) {
			float *in = buf + x * pass_stride + (y * stride + pass_offset.y) / sizeof(float);
			float *out = rgb + (x + y * sw) * 3 + (sw * sh) * 3;
			out[0] = in[0] / num_samples;
			out[1] = in[1] / num_samples;
			out[2] = in[2] / num_samples;
		}
		if (num_inputs > 2) {
			float *in = buf + x * pass_stride + (y * stride + pass_offset.z) / sizeof(float);
			float *out = rgb + (x + y * sw) * 3 + (sw * sh * 2) * 3;
			out[0] = in[0] / num_samples;
			out[1] = in[1] / num_samples;
			out[2] = in[2] / num_samples;
		}
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_convert_from_rgb(float *rgb, float *buf, int ix, int iy, int iw, int ih, int sx, int sy, int sw, int sh, int offset, int stride, int pass_stride, int num_samples)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < sw && y < sh) {
		float *in = rgb + ((ix + x) + (iy + y) * iw) * 3;
		float *out = buf + (offset + (sx + x) + (sy + y) * stride) * pass_stride;
		out[0] = in[0] * num_samples;
		out[1] = in[1] * num_samples;
		out[2] = in[2] * num_samples;
	}
}


extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_shadow(int sample,
                                 CCL_FILTER_TILE_INFO,
                                 float *unfilteredA,
                                 float *unfilteredB,
                                 float *sampleVariance,
                                 float *sampleVarianceV,
                                 float *bufferVariance,
                                 int4 prefilter_rect,
                                 int buffer_pass_stride,
                                 int buffer_denoising_offset)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_divide_shadow(sample,
		                            tile_info,
		                            x, y,
		                            unfilteredA,
		                            unfilteredB,
		                            sampleVariance,
		                            sampleVarianceV,
		                            bufferVariance,
		                            prefilter_rect,
		                            buffer_pass_stride,
		                            buffer_denoising_offset);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_get_feature(int sample,
                               CCL_FILTER_TILE_INFO,
                               int m_offset,
                               int v_offset,
                               float *mean,
                               float *variance,
                               float scale,
                               int4 prefilter_rect,
                               int buffer_pass_stride,
                               int buffer_denoising_offset)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_get_feature(sample,
		                          tile_info,
		                          m_offset, v_offset,
		                          x, y,
		                          mean, variance,
		                          scale,
		                          prefilter_rect,
		                          buffer_pass_stride,
		                          buffer_denoising_offset);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_write_feature(int sample,
                                 int4 buffer_params,
                                 int4 filter_area,
                                 float *from,
                                 float *buffer,
                                 int out_offset,
                                 int4 prefilter_rect)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		kernel_filter_write_feature(sample,
	                                x + filter_area.x,
	                                y + filter_area.y,
	                                buffer_params,
	                                from,
	                                buffer,
	                                out_offset,
	                                prefilter_rect);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_detect_outliers(float *image,
                                   float *variance,
                                   float *depth,
                                   float *output,
                                   int4 prefilter_rect,
                                   int pass_stride)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_detect_outliers(x, y, image, variance, depth, output, prefilter_rect, pass_stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_combine_halves(float *mean, float *variance, float *a, float *b, int4 prefilter_rect, int r)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_combine_halves(x, y, mean, variance, a, b, prefilter_rect, r);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_construct_transform(float const* __restrict__ buffer,
                                       CCL_FILTER_TILE_INFO,
                                       float *transform, int *rank,
                                       int4 filter_area, int4 rect,
                                       int radius, float pca_threshold,
                                       int pass_stride, int frame_stride,
                                       bool use_time)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int *l_rank = rank + y*filter_area.z + x;
		float *l_transform = transform + y*filter_area.z + x;
		kernel_filter_construct_transform(buffer,
		                                  tile_info,
		                                  x + filter_area.x, y + filter_area.y,
		                                  rect,
		                                  pass_stride, frame_stride,
		                                  use_time,
		                                  l_transform, l_rank,
		                                  radius, pca_threshold,
		                                  filter_area.z*filter_area.w,
		                                  threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_difference(const float *ccl_restrict weight_image,
                                       const float *ccl_restrict variance_image,
                                       const float *ccl_restrict scale_image,
                                       float *difference_image,
                                       int w,
                                       int h,
                                       int stride,
                                       int pass_stride,
                                       int r,
                                       int channel_offset,
                                       int frame_offset,
                                       float a,
                                       float k_2)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, pass_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_calc_difference(co.x, co.y, co.z, co.w,
		                                  weight_image,
		                                  variance_image,
		                                  scale_image,
		                                  difference_image + ofs,
		                                  rect, stride,
		                                  channel_offset,
		                                  frame_offset,
		                                  a, k_2);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_blur(const float *ccl_restrict difference_image,
                            float *out_image,
                            int w,
                            int h,
                            int stride,
                            int pass_stride,
                            int r,
                            int f)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, pass_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_blur(co.x, co.y,
		                       difference_image + ofs,
		                       out_image + ofs,
		                       rect, stride, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_weight(const float *ccl_restrict difference_image,
                                   float *out_image,
                                   int w,
                                   int h,
                                   int stride,
                                   int pass_stride,
                                   int r,
                                   int f)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, pass_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_calc_weight(co.x, co.y,
		                              difference_image + ofs,
		                              out_image + ofs,
		                              rect, stride, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_update_output(const float *ccl_restrict difference_image,
                                     const float *ccl_restrict image,
                                     float *out_image,
                                     float *accum_image,
                                     int w,
                                     int h,
                                     int stride,
                                     int pass_stride,
                                     int channel_offset,
                                     int r,
                                     int f)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords(w, h, r, pass_stride, &rect, &co, &ofs)) {
		kernel_filter_nlm_update_output(co.x, co.y, co.z, co.w,
		                                difference_image + ofs,
		                                image,
		                                out_image,
		                                accum_image,
		                                rect,
		                                channel_offset,
		                                stride, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_normalize(float *out_image,
                                 const float *ccl_restrict accum_image,
                                 int w,
                                 int h,
                                 int stride)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < w && y < h) {
		kernel_filter_nlm_normalize(x, y, out_image, accum_image, stride);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_construct_gramian(int t,
                                         const float *ccl_restrict difference_image,
                                         const float *ccl_restrict buffer,
                                         float const* __restrict__ transform,
                                         int *rank,
                                         float *XtWX,
                                         float3 *XtWY,
                                         int4 filter_window,
                                         int w,
                                         int h,
                                         int stride,
                                         int pass_stride,
                                         int r,
                                         int f,
                                         int frame_offset,
                                         bool use_time)
{
	int4 co, rect;
	int ofs;
	if(get_nlm_coords_window(w, h, r, pass_stride, &rect, &co, &ofs, filter_window)) {
		kernel_filter_nlm_construct_gramian(co.x, co.y,
		                                    co.z, co.w,
		                                    t,
		                                    difference_image + ofs,
		                                    buffer,
		                                    transform, rank,
		                                    XtWX, XtWY,
		                                    rect, filter_window,
		                                    stride, f,
		                                    pass_stride,
		                                    frame_offset,
		                                    use_time,
		                                    threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_finalize(float *buffer,
                            int *rank,
                            float *XtWX,
                            float3 *XtWY,
                            int4 filter_area,
                            int4 buffer_params,
                            int sample)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int storage_ofs = y*filter_area.z+x;
		rank += storage_ofs;
		XtWX += storage_ofs;
		XtWY += storage_ofs;
		kernel_filter_finalize(x, y, buffer, rank,
		                       filter_area.z*filter_area.w,
		                       XtWX, XtWY,
		                       buffer_params, sample);
	}
}

#endif

