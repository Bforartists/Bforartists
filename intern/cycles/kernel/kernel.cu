#include "hip/hip_runtime.h"
/*
 * Copyright 2011, Blender Foundation.
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301, USA.
 */

/* CUDA kernel entry points */

#include "kernel_compat_cuda.h"
#include "kernel_math.h"
#include "kernel_types.h"
#include "kernel_globals.h"
#include "kernel_film.h"
#include "kernel_path.h"
#include "kernel_displace.h"

extern "C" __global__ void kernel_cuda_path_trace(float4 *buffer, uint *rng_state, int pass, int sx, int sy, int sw, int sh)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_path_trace(NULL, buffer, rng_state, pass, x, y);
}

extern "C" __global__ void kernel_cuda_tonemap(uchar4 *rgba, float4 *buffer, int pass, int resolution, int sx, int sy, int sw, int sh)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_tonemap(NULL, rgba, buffer, pass, resolution, x, y);
}

extern "C" __global__ void kernel_cuda_displace(uint4 *input, float3 *offset, int sx)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	kernel_displace(NULL, input, offset, x);
}

