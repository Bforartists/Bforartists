#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License
 */

/* CUDA kernel entry points */

#include "kernel_compat_cuda.h"
#include "kernel_math.h"
#include "kernel_types.h"
#include "kernel_globals.h"
#include "kernel_film.h"
#include "kernel_path.h"
#include "kernel_displace.h"

extern "C" __global__ void kernel_cuda_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void kernel_cuda_branched_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_branched_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}
#endif

extern "C" __global__ void kernel_cuda_tonemap(uchar4 *rgba, float *buffer, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_tonemap(NULL, rgba, buffer, sample, x, y, offset, stride);
}

extern "C" __global__ void kernel_cuda_shader(uint4 *input, float4 *output, int type, int sx)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	kernel_shader_evaluate(NULL, input, output, (ShaderEvalType)type, x);
}

