#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <hip/hip_runtime.h>
#include "AdjacentMatrix.h"

__device__ __host__ glm::dvec3
middle_point(const glm::dvec3 &p0, const glm::dvec3 &n0, const glm::dvec3 &p1, const glm::dvec3 &n1) {
	/* How was this derived?
	*
	* Minimize \|x-p0\|^2 + \|x-p1\|^2, where
	* dot(n0, x) == dot(n0, p0)
	* dot(n1, x) == dot(n1, p1)
	*
	* -> Lagrange multipliers, set derivative = 0
	*  Use first 3 equalities to write x in terms of
	*  lambda_1 and lambda_2. Substitute that into the last
	*  two equations and solve for the lambdas. Finally,
	*  add a small epsilon term to avoid issues when n1=n2.
	*/
	double n0p0 = glm::dot(n0, p0), n0p1 = glm::dot(n0, p1),
		n1p0 = glm::dot(n1, p0), n1p1 = glm::dot(n1, p1),
		n0n1 = glm::dot(n0, n1),
		denom = 1.0f / (1.0f - n0n1*n0n1 + 1e-4f),
		lambda_0 = 2.0f*(n0p1 - n0p0 - n0n1*(n1p0 - n1p1))*denom,
		lambda_1 = 2.0f*(n1p0 - n1p1 - n0n1*(n0p1 - n0p0))*denom;

	return 0.5 * (p0 + p1) - 0.25 * (n0 * lambda_0 + n1 * lambda_1);
}

__device__ __host__ glm::dvec3
position_round_4(const  glm::dvec3 &o, const  glm::dvec3 &q,
const  glm::dvec3 &n, const  glm::dvec3 &p,
double scale) {
	double inv_scale = 1.0 / scale;
	glm::dvec3 t = glm::cross(n, q);
	glm::dvec3 d = p - o;
	return o +
		q * std::round(glm::dot(q, d) * inv_scale) * scale +
		t * std::round(glm::dot(t, d) * inv_scale) * scale;
}

__device__ __host__ glm::dvec3
position_floor_4(const glm::dvec3 &o, const glm::dvec3 &q,
const glm::dvec3 &n, const glm::dvec3 &p,
double scale) {
	double inv_scale = 1.0 / scale;
	glm::dvec3 t = glm::cross(n,q);
	glm::dvec3 d = p - o;
	return o +
		q * std::floor(glm::dot(q, d) * inv_scale) * scale +
		t * std::floor(glm::dot(t, d) * inv_scale) * scale;
}


__device__ __host__ double cudaSignum(double value) {
	return std::copysign((double)1, value);
}

__device__ __host__ void
compat_orientation_extrinsic_4(const glm::dvec3 &q0, const glm::dvec3 &n0,
const glm::dvec3 &q1, const glm::dvec3 &n1, glm::dvec3& value1, glm::dvec3& value2) {
	const glm::dvec3 A[2] = { q0, glm::cross(n0, q0) };
	const glm::dvec3 B[2] = { q1, glm::cross(n1, q1) };

	double best_score = -1e10;
	int best_a = 0, best_b = 0;

	for (int i = 0; i < 2; ++i) {
		for (int j = 0; j < 2; ++j) {
			double score = std::abs(glm::dot(A[i], B[j]));
			if (score > best_score + 1e-6) {
				best_a = i;
				best_b = j;
				best_score = score;
			}
		}
	}
	const double dp = glm::dot(A[best_a], B[best_b]);
	value1 = A[best_a];
	value2 = B[best_b] * cudaSignum(dp);
}

__device__ __host__ void
compat_position_extrinsic_4(
const glm::dvec3 &p0, const glm::dvec3 &n0, const glm::dvec3 &q0, const glm::dvec3 &o0,
const glm::dvec3 &p1, const glm::dvec3 &n1, const glm::dvec3 &q1, const glm::dvec3 &o1,
double scale, glm::dvec3& v1, glm::dvec3& v2) {

	glm::dvec3 t0 = glm::cross(n0, q0), t1 = glm::cross(n1, q1);
	glm::dvec3 middle = middle_point(p0, n0, p1, n1);
	glm::dvec3 o0p = position_floor_4(o0, q0, n0, middle, scale);
	glm::dvec3 o1p = position_floor_4(o1, q1, n1, middle, scale);

	double best_cost = 1e10;
	int best_i = -1, best_j = -1;

	for (int i = 0; i<4; ++i) {
		glm::dvec3 o0t = o0p + (q0 * ((i & 1) * scale) + t0 * (((i & 2) >> 1) * scale));
		for (int j = 0; j<4; ++j) {
			glm::dvec3 o1t = o1p + (q1 * ((j & 1) * scale) + t1 * (((j & 2) >> 1) * scale));
			glm::dvec3 t = o0t - o1t;
			double cost = glm::dot(t, t);

			if (cost < best_cost) {
				best_i = i;
				best_j = j;
				best_cost = cost;
			}
		}
	}

	v1 = o0p + (q0 * ((best_i & 1) * scale) + t0 * (((best_i & 2) >> 1) * scale)),
	v2 = o1p + (q1 * ((best_j & 1) * scale) + t1 * (((best_j & 2) >> 1) * scale));
}

__global__ 
void cudaUpdateOrientation(int* phase, int num_phases, glm::dvec3* N, glm::dvec3* Q, Link* adj, int* adjOffset, int num_adj) {
	int pi = blockIdx.x * blockDim.x + threadIdx.x;

//	for (int pi = 0; pi < num_phases; ++pi) {
		if (pi >= num_phases)
			return;
		int i = phase[pi];
		glm::dvec3 n_i = N[i];
		double weight_sum = 0.0f;
		glm::dvec3 sum = Q[i];

		for (int l = adjOffset[i]; l < adjOffset[i + 1]; ++l) {
			Link link = adj[l];
			const int j = link.id;
			const double weight = link.weight;
			if (weight == 0)
				continue;
			glm::dvec3 n_j = N[j];
			glm::dvec3 q_j = Q[j];
			glm::dvec3 value1, value2;
			compat_orientation_extrinsic_4(sum, n_i, q_j, n_j, value1, value2);
			sum = value1 * weight_sum + value2 * weight;
			sum -= n_i*glm::dot(n_i, sum);
			weight_sum += weight;

			double norm = glm::length(sum);
			if (norm > 2.93873587705571876e-39f)
				sum /= norm;
		}

		if (weight_sum > 0) {
			Q[i] = sum;
		}
//	}
}

__global__
void cudaPropagateOrientationUpper(glm::dvec3* srcField, glm::ivec2* toUpper, glm::dvec3* N, glm::dvec3* destField, int num_orientation) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int i = 0; i < num_orientation; ++i) {
		if (i >= num_orientation)
			return;
		for (int k = 0; k < 2; ++k) {
			int dest = toUpper[i][k];
			if (dest == -1)
				continue;
			glm::dvec3 q = srcField[i];
			glm::dvec3 n = N[dest];
			destField[dest] = q - n * glm::dot(n, q);
		}
//	}
}

__global__
void cudaPropagateOrientationLower(glm::ivec2* toUpper, glm::dvec3* Q, glm::dvec3* N, glm::dvec3* Q_next, glm::dvec3* N_next, int num_toUpper) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int i = 0; i < num_toUpper; ++i) {
		if (i >= num_toUpper)
			return;
		glm::ivec2 upper = toUpper[i];
		glm::dvec3 q0 = Q[upper[0]];
		glm::dvec3 n0 = N[upper[0]];

		glm::dvec3 q, q1, n1, value1, value2;
		if (upper[1] != -1) {
			q1 = Q[upper[1]];
			n1 = N[upper[1]];
			compat_orientation_extrinsic_4(q0, n0, q1, n1, value1, value2);
			q = value1 + value2;
		}
		else {
			q = q0;
		}
		glm::dvec3 n = N_next[i];
		q -= glm::dot(n, q) * n;

		double len = q.x * q.x + q.y * q.y + q.z * q.z;
		if (len > 2.93873587705571876e-39f)
			q /= sqrt(len);
		Q_next[i] = q;
//	}
}


__global__ 
void cudaUpdatePosition(int* phase, int num_phases, glm::dvec3* N, glm::dvec3* Q, Link* adj, int* adjOffset, int num_adj, glm::dvec3* V, glm::dvec3* O, double scale) {
	int pi = blockIdx.x * blockDim.x + threadIdx.x;

//	for (int pi = 0; pi < num_phases; ++pi) {
	if (pi >= num_phases)
		return;
		int i = phase[pi];
		glm::dvec3 n_i = N[i], v_i = V[i];
		glm::dvec3 q_i = Q[i];
		glm::dvec3 sum = O[i];
		double weight_sum = 0.0f;

		for (int l = adjOffset[i]; l < adjOffset[i + 1]; ++l) {
			Link link = adj[l];
			int j = link.id;
			const double weight = link.weight;
			if (weight == 0)
				continue;

			glm::dvec3 n_j = N[j], v_j = V[j];
			glm::dvec3 q_j = Q[j], o_j = O[j];
			glm::dvec3 v1, v2;
			compat_position_extrinsic_4(
				v_i, n_i, q_i, sum, v_j, n_j, q_j, o_j, scale, v1, v2);

			sum = v1*weight_sum +v2*weight;
			weight_sum += weight;
			if (weight_sum > 2.93873587705571876e-39f)
				sum /= weight_sum;
			sum -= glm::dot(n_i, sum - v_i)*n_i;
		}

		if (weight_sum > 0) {
			O[i] = position_round_4(sum, q_i, n_i, v_i, scale);
		}
//	}
}

__global__
void cudaPropagatePositionUpper(glm::dvec3* srcField, glm::ivec2* toUpper, glm::dvec3* N, glm::dvec3* V, glm::dvec3* destField, int num_position) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int i = 0; i < num_position; ++i) {
	if (i >= num_position)
		return;
		for (int k = 0; k < 2; ++k) {
			int dest = toUpper[i][k];
			if (dest == -1)
				continue;
			glm::dvec3 o = srcField[i], n = N[dest], v = V[dest];
			o -= n * glm::dot(n, o - v);
			destField[dest] = o;
		}
//	}
}


void UpdateOrientation(int* phase, int num_phases, glm::dvec3* N, glm::dvec3* Q, Link* adj, int* adjOffset, int num_adj) {
	cudaUpdateOrientation << <(num_phases + 255) / 256, 256 >> >(phase, num_phases, N, Q, adj, adjOffset, num_adj);
//	cudaUpdateOrientation(phase, num_phases, N, Q, adj, adjOffset, num_adj);
}

void PropagateOrientationUpper(glm::dvec3* srcField, int num_orientation, glm::ivec2* toUpper, glm::dvec3* N, glm::dvec3* destField) {
	cudaPropagateOrientationUpper << <(num_orientation + 255) / 256, 256 >> >(srcField, toUpper, N, destField, num_orientation);
//	cudaPropagateOrientationUpper(srcField, toUpper, N, destField, num_orientation);
}

void PropagateOrientationLower(glm::ivec2* toUpper, glm::dvec3* Q, glm::dvec3* N, glm::dvec3* Q_next, glm::dvec3* N_next, int num_toUpper) {
	cudaPropagateOrientationLower << <(num_toUpper + 255) / 256, 256 >> >(toUpper, Q, N, Q_next, N_next, num_toUpper);
//	cudaPropagateOrientationLower(toUpper, Q, N, Q_next, N_next, num_toUpper);
}


void UpdatePosition(int* phase, int num_phases, glm::dvec3* N, glm::dvec3* Q, Link* adj, int* adjOffset, int num_adj, glm::dvec3* V, glm::dvec3* O, double scale) {
	cudaUpdatePosition << <(num_phases + 255) / 256, 256 >> >(phase, num_phases, N, Q, adj, adjOffset, num_adj, V, O, scale);
//	cudaUpdatePosition(phase, num_phases, N, Q, adj, adjOffset, num_adj, V, O, scale);
}

void PropagatePositionUpper(glm::dvec3* srcField, int num_position, glm::ivec2* toUpper, glm::dvec3* N, glm::dvec3* V, glm::dvec3* destField) {
	cudaPropagatePositionUpper << <(num_position + 255) / 256, 256 >> >(srcField, toUpper, N, V, destField, num_position);
//	cudaPropagatePositionUpper(srcField, toUpper, N, V, destField, num_position);
}
